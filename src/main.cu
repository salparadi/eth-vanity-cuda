#include "hip/hip_runtime.h"
/*
    Copyright (C) 2025 Manuel

    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU Affero General Public License as published by
    the Free Software Foundation, version 3.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU Affero General Public License for more details.

    You should have received a copy of the GNU Affero General Public License
    along with this program.  If not, see <https://www.gnu.org/licenses/>.
*/

#if defined(_WIN64)
    #define WIN32_NO_STATUS
    #include <windows.h>
    #undef WIN32_NO_STATUS
#endif

#include <thread>
#include <cinttypes>
#include <iomanip>
#include <iostream>
#include <mutex>
#include <queue>
#include <chrono>
#include <fstream>
#include <vector>
#include <array>
#include <algorithm>
#include <cstring>
#include <ctype.h>
#include <stdio.h>

#include "secure_rand.h"
#include "structures.h"

#include "cpu_curve_math.h"
#include "cpu_keccak.h"
#include "cpu_math.h"


#define OUTPUT_BUFFER_SIZE 10000

#define BLOCK_SIZE 256U
#define THREAD_WORK (1U << 8)



__constant__ CurvePoint thread_offsets[BLOCK_SIZE];
__constant__ CurvePoint addends[THREAD_WORK - 1];
__device__ uint64_t device_memory[2 + OUTPUT_BUFFER_SIZE * 3];

__device__ int count_zero_bytes(uint32_t x) {
    int n = 0;
    n += ((x & 0xFF) == 0);
    n += ((x & 0xFF00) == 0);
    n += ((x & 0xFF0000) == 0);
    n += ((x & 0xFF000000) == 0);
    return n;
}

__device__ int score_zero_bytes(Address a) {
    int n = 0;
    n += count_zero_bytes(a.a);
    n += count_zero_bytes(a.b);
    n += count_zero_bytes(a.c);
    n += count_zero_bytes(a.d);
    n += count_zero_bytes(a.e);
    return n;
}

__device__ int score_leading_zeros(Address a) {
    int n = __clz(a.a);
    if (n == 32) {
        n += __clz(a.b);

        if (n == 64) {
            n += __clz(a.c);

            if (n == 96) {
                n += __clz(a.d);

                if (n == 128) {
                    n += __clz(a.e);
                }
            }
        }
    }

    return n >> 3;
}

#ifdef __linux__
    #define atomicMax_ul(a, b) atomicMax((unsigned long long*)(a), (unsigned long long)(b))
    #define atomicAdd_ul(a, b) atomicAdd((unsigned long long*)(a), (unsigned long long)(b))
#else
    #define atomicMax_ul(a, b) atomicMax(a, b)
    #define atomicAdd_ul(a, b) atomicAdd(a, b)
#endif

__device__ int score_prefix_suffix(Address a, const char* prefix, int prefix_len, const char* suffix, int suffix_len) {
    // DEBUG: print prefix_len and suffix_len when called
    // printf("[DEBUG] score_prefix_suffix called with prefix_len=%d, suffix_len=%d\n", prefix_len, suffix_len);

    // Convert Address to hex string (40 chars)
    char hex[41];
    #pragma unroll
    for (int i = 0; i < 5; i++) {
        uint32_t val;
        if (i == 0) val = a.a;
        else if (i == 1) val = a.b;
        else if (i == 2) val = a.c;
        else if (i == 3) val = a.d;
        else val = a.e;
        // Write 8 hex chars for each uint32
        #pragma unroll
        for (int j = 0; j < 8; j++) {
            int shift = 28 - j * 4;
            int nibble = (val >> shift) & 0xF;
            char c = (nibble < 10) ? ('0' + nibble) : ('a' + nibble - 10);
            hex[i * 8 + j] = c;
        }
    }
    hex[40] = '\0';
    // Check prefix
    bool prefix_match = true;
    for (int i = 0; i < prefix_len; i++) {
        if (hex[i] != prefix[i]) {
            prefix_match = false;
            break;
        }
    }
    if (!prefix_match) return 0;
    // Check suffix
    bool suffix_match = true;
    for (int i = 0; i < suffix_len; i++) {
        if (hex[40 - suffix_len + i] != suffix[i]) {
            suffix_match = false;
            break;
        }
    }
    if (!suffix_match) return 0;
    return 1;
}


// Prefix/suffix for address matching (copied to device constant memory)
__constant__ char device_prefix[64];
__constant__ char device_suffix[64];

__device__ void handle_output(int score_method, Address a, uint64_t key, bool inv) {
    int score = 0;
    if (score_method == 0) { score = score_leading_zeros(a); }
    else if (score_method == 1) { score = score_zero_bytes(a); }
    else if (score_method == 2) {
        // printf("Brute-forcing for Prefix & Suffix");
        // Use device_prefix/device_suffix and their actual lengths
        int prefix_len = 0;
        int suffix_len = 0;
        while (prefix_len < 64 && device_prefix[prefix_len] != '\0') prefix_len++;
        while (suffix_len < 64 && device_suffix[suffix_len] != '\0') suffix_len++;
        score = score_prefix_suffix(a, device_prefix, prefix_len, device_suffix, suffix_len);
        // Only push if score > 0
        if (score > 0) {
            uint32_t idx = atomicAdd_ul(&device_memory[0], 1);
            if (idx < OUTPUT_BUFFER_SIZE) {
                device_memory[2 + idx] = key;
                device_memory[OUTPUT_BUFFER_SIZE + 2 + idx] = score;
                device_memory[OUTPUT_BUFFER_SIZE * 2 + 2 + idx] = inv;
            }
        }
        return;
    }

    if (score >= device_memory[1]) {
        atomicMax_ul(&device_memory[1], score);
        if (score >= device_memory[1]) {
            uint32_t idx = atomicAdd_ul(&device_memory[0], 1);
            if (idx < OUTPUT_BUFFER_SIZE) {
                device_memory[2 + idx] = key;
                device_memory[OUTPUT_BUFFER_SIZE + 2 + idx] = score;
                device_memory[OUTPUT_BUFFER_SIZE * 2 + 2 + idx] = inv;
            }
        }
    }
}

__device__ void handle_output2(int score_method, Address a, uint64_t key) {
    int score = 0;
    if (score_method == 0) { score = score_leading_zeros(a); }
    else if (score_method == 1) { score = score_zero_bytes(a); }
    else if (score_method == 2) {
        int prefix_len = 0;
        int suffix_len = 0;
        while (prefix_len < 64 && device_prefix[prefix_len] != '\0') prefix_len++;
        while (suffix_len < 64 && device_suffix[suffix_len] != '\0') suffix_len++;
        score = score_prefix_suffix(a, device_prefix, prefix_len, device_suffix, suffix_len);
        // Only push if score > 0
        if (score > 0) {
            uint32_t idx = atomicAdd_ul(&device_memory[0], 1);
            if (idx < OUTPUT_BUFFER_SIZE) {
                device_memory[2 + idx] = key;
                device_memory[OUTPUT_BUFFER_SIZE + 2 + idx] = score;
            }
        }
        return;
    }

    if (score >= device_memory[1]) {
        atomicMax_ul(&device_memory[1], score);
        if (score >= device_memory[1]) {
            uint32_t idx = atomicAdd_ul(&device_memory[0], 1);
            if (idx < OUTPUT_BUFFER_SIZE) {
                device_memory[2 + idx] = key;
                device_memory[OUTPUT_BUFFER_SIZE + 2 + idx] = score;
            }
        }
    }
}

#include "address.h"
#include "contract_address.h"
#include "contract_address2.h"
#include "contract_address3.h"


int global_max_score = 0;
std::mutex global_max_score_mutex;
uint32_t GRID_SIZE = 1U << 15;

struct Message {
    uint64_t time;

    int status;
    int device_index;
    hipError_t error;

    double speed;
    int results_count;
    _uint256* results;
    int* scores;
};

std::queue<Message> message_queue;
std::mutex message_queue_mutex;


#define gpu_assert(call) { \
    hipError_t e = call; \
    if (e != hipSuccess) { \
        message_queue_mutex.lock(); \
        message_queue.push(Message{milliseconds(), 1, device_index, e}); \
        message_queue_mutex.unlock(); \
        if (thread_offsets_host != 0) { hipHostFree(thread_offsets_host); } \
        if (device_memory_host != 0) { hipHostFree(device_memory_host); } \
        hipDeviceReset(); \
        return; \
    } \
}

uint64_t milliseconds() {
    return (std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::system_clock::now().time_since_epoch())).count();
}


void host_thread(int device, int device_index, int score_method, int mode, Address origin_address, Address deployer_address, _uint256 bytecode, const char* prefix, const char* suffix) {
    const uint32_t grid_size_clamped = std::max<uint32_t>(1U, GRID_SIZE);
    const uint64_t GRID_WORK = static_cast<uint64_t>(BLOCK_SIZE) * static_cast<uint64_t>(grid_size_clamped) * static_cast<uint64_t>(THREAD_WORK);

    CurvePoint* block_offsets = 0;
    CurvePoint* offsets = 0;
    CurvePoint* thread_offsets_host = 0;

    uint64_t* device_memory_host = 0;
    uint64_t* max_score_host;
    uint64_t* output_counter_host;
    uint64_t* output_buffer_host;
    uint64_t* output_buffer2_host;
    uint64_t* output_buffer3_host;

    gpu_assert(hipSetDevice(device));
    gpu_assert(hipMemcpyToSymbol(HIP_SYMBOL(device_prefix), prefix, 64, 0, hipMemcpyHostToDevice));
    gpu_assert(hipMemcpyToSymbol(HIP_SYMBOL(device_suffix), suffix, 64, 0, hipMemcpyHostToDevice));

    gpu_assert(hipHostAlloc(&device_memory_host, (2 + OUTPUT_BUFFER_SIZE * 3) * sizeof(uint64_t), hipHostMallocDefault))
    output_counter_host = device_memory_host;
    max_score_host = device_memory_host + 1;
    output_buffer_host = max_score_host + 1;
    output_buffer2_host = output_buffer_host + OUTPUT_BUFFER_SIZE;
    output_buffer3_host = output_buffer2_host + OUTPUT_BUFFER_SIZE;

    output_counter_host[0] = 0;
    max_score_host[0] = (score_method == 2) ? 1 : 2;
    gpu_assert(hipMemcpyToSymbol(HIP_SYMBOL(device_memory), device_memory_host, 2 * sizeof(uint64_t)));
    gpu_assert(hipDeviceSynchronize())


    if (mode == 0 || mode == 1) {
        gpu_assert(hipMalloc(&block_offsets, grid_size_clamped * sizeof(CurvePoint)))
        gpu_assert(hipMalloc(&offsets, static_cast<uint64_t>(grid_size_clamped) * BLOCK_SIZE * sizeof(CurvePoint)))
        gpu_assert(hipHostAlloc(&thread_offsets_host, BLOCK_SIZE * sizeof(CurvePoint), hipHostMallocWriteCombined))
    }

    _uint256 max_key;
    if (mode == 0 || mode == 1) {
        _uint256 GRID_WORK = cpu_mul_256_mod_p(cpu_mul_256_mod_p(_uint256{0, 0, 0, 0, 0, 0, 0, THREAD_WORK}, _uint256{0, 0, 0, 0, 0, 0, 0, BLOCK_SIZE}), _uint256{0, 0, 0, 0, 0, 0, 0, grid_size_clamped});
        max_key = _uint256{0x7FFFFFFF, 0xFFFFFFFF, 0xFFFFFFFF, 0xFFFFFFFF, 0x5D576E73, 0x57A4501D, 0xDFE92F46, 0x681B20A0};
        max_key = cpu_sub_256(max_key, GRID_WORK);
        max_key = cpu_sub_256(max_key, _uint256{0, 0, 0, 0, 0, 0, 0, THREAD_WORK});
        max_key = cpu_add_256(max_key, _uint256{0, 0, 0, 0, 0, 0, 0, 2});
    } else if (mode == 2 || mode == 3) {
        max_key = _uint256{0xFFFFFFFF, 0xFFFFFFFF, 0xFFFFFFFF, 0xFFFFFFFF, 0xFFFFFFFF, 0xFFFFFFFF, 0xFFFFFFFF, 0xFFFFFFFF};
    }

    _uint256 base_random_key{0, 0, 0, 0, 0, 0, 0, 0};
    _uint256 random_key_increment{0, 0, 0, 0, 0, 0, 0, 0};
    int status;
    if (mode == 0 || mode == 1) {
        status = generate_secure_random_key(base_random_key, max_key, 255);
        random_key_increment = cpu_mul_256_mod_p(cpu_mul_256_mod_p(uint32_to_uint256(BLOCK_SIZE), uint32_to_uint256(grid_size_clamped)), uint32_to_uint256(THREAD_WORK));
    } else if (mode == 2 || mode == 3) {
        status = generate_secure_random_key(base_random_key, max_key, 256);
        random_key_increment = cpu_mul_256_mod_p(cpu_mul_256_mod_p(uint32_to_uint256(BLOCK_SIZE), uint32_to_uint256(grid_size_clamped)), uint32_to_uint256(THREAD_WORK));
        base_random_key.h &= ~(THREAD_WORK - 1);
    }

    if (status) {
        message_queue_mutex.lock();
        message_queue.push(Message{milliseconds(), 10 + status, device_index});
        message_queue_mutex.unlock();
        return;
    }
    _uint256 random_key = base_random_key;

    if (mode == 0 || mode == 1) {
        CurvePoint* addends_host = new CurvePoint[THREAD_WORK - 1];
        CurvePoint p = G;
        for (int i = 0; i < THREAD_WORK - 1; i++) {
            addends_host[i] = p;
            p = cpu_point_add(p, G);
        }
        gpu_assert(hipMemcpyToSymbol(HIP_SYMBOL(addends), addends_host, (THREAD_WORK - 1) * sizeof(CurvePoint)))
        delete[] addends_host;

        CurvePoint* block_offsets_host = new CurvePoint[grid_size_clamped];
        CurvePoint block_offset = cpu_point_multiply(G, _uint256{0, 0, 0, 0, 0, 0, 0, THREAD_WORK * BLOCK_SIZE});
        p = G;
        for (uint32_t i = 0; i < grid_size_clamped; i++) {
            block_offsets_host[i] = p;
            p = cpu_point_add(p, block_offset);
        }
        gpu_assert(hipMemcpy(block_offsets, block_offsets_host, grid_size_clamped * sizeof(CurvePoint), hipMemcpyHostToDevice))
        delete[] block_offsets_host;
    }

    if (mode == 0 || mode == 1) {
        hipStream_t streams[2];
        gpu_assert(hipStreamCreate(&streams[0]))
        gpu_assert(hipStreamCreate(&streams[1]))
        
        _uint256 previous_random_key = random_key;
        bool first_iteration = true;
        uint64_t start_time;
        uint64_t end_time;
        double elapsed;

        while (true) {
            if (!first_iteration) {
                if (mode == 0) {
                    gpu_address_work<<<grid_size_clamped, BLOCK_SIZE, 0, streams[0]>>>(score_method, offsets);
                } else {
                    gpu_contract_address_work<<<grid_size_clamped, BLOCK_SIZE, 0, streams[0]>>>(score_method, offsets);
                }
            }

            if (!first_iteration) {
                previous_random_key = random_key;
                random_key = cpu_add_256(random_key, random_key_increment);
                if (gte_256(random_key, max_key)) {
                    random_key = cpu_sub_256(random_key, max_key);
                }
            }
            CurvePoint thread_offset = cpu_point_multiply(G, _uint256{0, 0, 0, 0, 0, 0, 0, THREAD_WORK});
            CurvePoint p = cpu_point_multiply(G, cpu_add_256(_uint256{0, 0, 0, 0, 0, 0, 0, THREAD_WORK - 1}, random_key));
            for (int i = 0; i < BLOCK_SIZE; i++) {
                thread_offsets_host[i] = p;
                p = cpu_point_add(p, thread_offset);
            }
            gpu_assert(hipMemcpyToSymbolAsync(HIP_SYMBOL(thread_offsets), thread_offsets_host, BLOCK_SIZE * sizeof(CurvePoint), 0, hipMemcpyHostToDevice, streams[1]));
            gpu_assert(hipStreamSynchronize(streams[1]))
            gpu_assert(hipStreamSynchronize(streams[0]))

            if (!first_iteration) {
                end_time = milliseconds();
                elapsed = (end_time - start_time) / 1000.0;
            }
            start_time = milliseconds();

            const uint32_t init_grid = std::max<uint32_t>(1U, (grid_size_clamped + BLOCK_SIZE - 1) / BLOCK_SIZE);
            gpu_address_init<<<init_grid, BLOCK_SIZE, 0, streams[0]>>>(block_offsets, offsets, grid_size_clamped);
            if (!first_iteration) {
                gpu_assert(hipMemcpyFromSymbolAsync(device_memory_host, HIP_SYMBOL(device_memory), (2 + OUTPUT_BUFFER_SIZE * 3) * sizeof(uint64_t), 0, hipMemcpyDeviceToHost, streams[1]))
                gpu_assert(hipStreamSynchronize(streams[1]))
            }
            if (!first_iteration) {
                global_max_score_mutex.lock();
                if (output_counter_host[0] != 0) {
                    if (max_score_host[0] > global_max_score) {
                        global_max_score = max_score_host[0];
                    } else {
                        max_score_host[0] = global_max_score;
                    }
                }
                global_max_score_mutex.unlock();

                double speed = GRID_WORK / elapsed / 1000000.0 * 2;
                if (output_counter_host[0] != 0) {
                    int valid_results = 0;

                    for (int i = 0; i < output_counter_host[0]; i++) {
                        if (output_buffer2_host[i] < max_score_host[0]) { continue; }
                        valid_results++;
                    }

                    if (valid_results > 0) {
                        _uint256* results = new _uint256[valid_results];
                        int* scores = new int[valid_results];
                        valid_results = 0;

                        for (int i = 0; i < output_counter_host[0]; i++) {
                            if (output_buffer2_host[i] < max_score_host[0]) { continue; }

                            uint64_t k_offset = output_buffer_host[i];
                            _uint256 k = cpu_add_256(previous_random_key, cpu_add_256(_uint256{0, 0, 0, 0, 0, 0, 0, THREAD_WORK}, _uint256{0, 0, 0, 0, 0, 0, (uint32_t)(k_offset >> 32), (uint32_t)(k_offset & 0xFFFFFFFF)}));

                            if (output_buffer3_host[i]) {
                                k = cpu_sub_256(N, k);
                            }
                
                            int idx = valid_results++;
                            results[idx] = k;
                            scores[idx] = output_buffer2_host[i];
                        }

                        message_queue_mutex.lock();
                        message_queue.push(Message{end_time, 0, device_index, hipSuccess, speed, valid_results, results, scores});
                        message_queue_mutex.unlock();
                    } else {
                        message_queue_mutex.lock();
                        message_queue.push(Message{end_time, 0, device_index, hipSuccess, speed, 0});
                        message_queue_mutex.unlock();
                    }
                } else {
                    message_queue_mutex.lock();
                    message_queue.push(Message{end_time, 0, device_index, hipSuccess, speed, 0});
                    message_queue_mutex.unlock();
                }
            }

            if (!first_iteration) {
                output_counter_host[0] = 0;
                gpu_assert(hipMemcpyToSymbolAsync(HIP_SYMBOL(device_memory), device_memory_host, sizeof(uint64_t), 0, hipMemcpyHostToDevice, streams[1]));
                gpu_assert(hipStreamSynchronize(streams[1]))
            }
            gpu_assert(hipStreamSynchronize(streams[0]))
            first_iteration = false;
        }
    }

    if (mode == 2) {
        while (true) {
            uint64_t start_time = milliseconds();
            gpu_contract2_address_work<<<grid_size_clamped, BLOCK_SIZE>>>(score_method, origin_address, random_key, bytecode);

            gpu_assert(hipDeviceSynchronize())
            gpu_assert(hipMemcpyFromSymbol(device_memory_host, HIP_SYMBOL(device_memory), (2 + OUTPUT_BUFFER_SIZE * 3) * sizeof(uint64_t)))

            uint64_t end_time = milliseconds();
            double elapsed = (end_time - start_time) / 1000.0;

            global_max_score_mutex.lock();
            if (output_counter_host[0] != 0) {
                if (max_score_host[0] > global_max_score) {
                    global_max_score = max_score_host[0];
                } else {
                    max_score_host[0] = global_max_score;
                }
            }
            global_max_score_mutex.unlock();

            double speed = GRID_WORK / elapsed / 1000000.0;
            if (output_counter_host[0] != 0) {
                int valid_results = 0;

                for (int i = 0; i < output_counter_host[0]; i++) {
                    if (output_buffer2_host[i] < max_score_host[0]) { continue; }
                    valid_results++;
                }

                if (valid_results > 0) {
                    _uint256* results = new _uint256[valid_results];
                    int* scores = new int[valid_results];
                    valid_results = 0;

                    for (int i = 0; i < output_counter_host[0]; i++) {
                        if (output_buffer2_host[i] < max_score_host[0]) { continue; }

                        uint64_t k_offset = output_buffer_host[i];
                        _uint256 k = cpu_add_256(random_key, _uint256{0, 0, 0, 0, 0, 0, (uint32_t)(k_offset >> 32), (uint32_t)(k_offset & 0xFFFFFFFF)});
            
                        int idx = valid_results++;
                        results[idx] = k;
                        scores[idx] = output_buffer2_host[i];
                    }

                    message_queue_mutex.lock();
                    message_queue.push(Message{end_time, 0, device_index, hipSuccess, speed, valid_results, results, scores});
                    message_queue_mutex.unlock();
                } else {
                    message_queue_mutex.lock();
                    message_queue.push(Message{end_time, 0, device_index, hipSuccess, speed, 0});
                    message_queue_mutex.unlock();
                }
            } else {
                message_queue_mutex.lock();
                message_queue.push(Message{end_time, 0, device_index, hipSuccess, speed, 0});
                message_queue_mutex.unlock();
            }

            random_key = cpu_add_256(random_key, random_key_increment);

            output_counter_host[0] = 0;
            gpu_assert(hipMemcpyToSymbol(HIP_SYMBOL(device_memory), device_memory_host, sizeof(uint64_t)));
        }
    }

    if (mode == 3) {
        while (true) {
            uint64_t start_time = milliseconds();
            gpu_contract3_address_work<<<grid_size_clamped, BLOCK_SIZE>>>(score_method, origin_address, deployer_address, random_key, bytecode);

            gpu_assert(hipDeviceSynchronize())
            gpu_assert(hipMemcpyFromSymbol(device_memory_host, HIP_SYMBOL(device_memory), (2 + OUTPUT_BUFFER_SIZE * 3) * sizeof(uint64_t)))

            uint64_t end_time = milliseconds();
            double elapsed = (end_time - start_time) / 1000.0;

            global_max_score_mutex.lock();
            if (output_counter_host[0] != 0) {
                if (max_score_host[0] > global_max_score) {
                    global_max_score = max_score_host[0];
                } else {
                    max_score_host[0] = global_max_score;
                }
            }
            global_max_score_mutex.unlock();

            double speed = GRID_WORK / elapsed / 1000000.0;
            if (output_counter_host[0] != 0) {
                int valid_results = 0;

                for (int i = 0; i < output_counter_host[0]; i++) {
                    if (output_buffer2_host[i] < max_score_host[0]) { continue; }
                    valid_results++;
                }

                if (valid_results > 0) {
                    _uint256* results = new _uint256[valid_results];
                    int* scores = new int[valid_results];
                    valid_results = 0;

                    for (int i = 0; i < output_counter_host[0]; i++) {
                        if (output_buffer2_host[i] < max_score_host[0]) { continue; }

                        uint64_t k_offset = output_buffer_host[i];
                        _uint256 k = cpu_add_256(random_key, _uint256{0, 0, 0, 0, 0, 0, (uint32_t)(k_offset >> 32), (uint32_t)(k_offset & 0xFFFFFFFF)});
            
                        int idx = valid_results++;
                        results[idx] = k;
                        scores[idx] = output_buffer2_host[i];
                    }

                    message_queue_mutex.lock();
                    message_queue.push(Message{end_time, 0, device_index, hipSuccess, speed, valid_results, results, scores});
                    message_queue_mutex.unlock();
                } else {
                    message_queue_mutex.lock();
                    message_queue.push(Message{end_time, 0, device_index, hipSuccess, speed, 0});
                    message_queue_mutex.unlock();
                }
            } else {
                message_queue_mutex.lock();
                message_queue.push(Message{end_time, 0, device_index, hipSuccess, speed, 0});
                message_queue_mutex.unlock();
            }

            random_key = cpu_add_256(random_key, random_key_increment);

            output_counter_host[0] = 0;
            gpu_assert(hipMemcpyToSymbol(HIP_SYMBOL(device_memory), device_memory_host, sizeof(uint64_t)));
        }
    }
}


void print_speeds(int num_devices, int* device_ids, const std::vector<double>& speeds) {

    double total = 0.0;
    for (int i = 0; i < num_devices; i++) {
        total += speeds[i];
    }

    printf("Total: %.2fM/s", total);
    for (int i = 0; i < num_devices; i++) {
        printf("  DEVICE %d: %.2fM/s", device_ids[i], speeds[i]);
    }
}


int main(int argc, char *argv[]) {
    int score_method = -1; // 0 = leading zeroes, 1 = zeros, 2 = prefix/suffix
    int mode = 0; // 0 = address, 1 = contract, 2 = create2 contract, 3 = create3 proxy contract
    char* input_file = 0;
    char* input_address = 0;
    char* input_deployer_address = 0;
    char* input_prefix = 0;
    char* input_suffix = 0;

    int num_devices = 0;
    int device_ids[10];

    for (int i = 1; i < argc;) {
        if (strcmp(argv[i], "--device") == 0 || strcmp(argv[i], "-d") == 0) {
            device_ids[num_devices++] = atoi(argv[i + 1]);
            i += 2;
        } else if (strcmp(argv[i], "--leading-zeros") == 0 || strcmp(argv[i], "-lz") == 0) {
            score_method = 0;
            i++;
        } else if (strcmp(argv[i], "--zeros") == 0 || strcmp(argv[i], "-z") == 0) {
            score_method = 1;
            i++;
        } else if (strcmp(argv[i], "--contract") == 0 || strcmp(argv[i], "-c") == 0) {
            mode = 1;
            i++;
        } else if (strcmp(argv[i], "--contract2") == 0 || strcmp(argv[i], "-c2") == 0) {
            mode = 2;
            i++;
        } else if (strcmp(argv[i], "--contract3") == 0 || strcmp(argv[i], "-c3") == 0) {
            mode = 3;
            i++;
        } else if (strcmp(argv[i], "--bytecode") == 0 || strcmp(argv[i], "-b") == 0) {
            input_file = argv[i + 1];
            i += 2;
        } else if  (strcmp(argv[i], "--address") == 0 || strcmp(argv[i], "-a") == 0) {
            input_address = argv[i + 1];
            i += 2;
        } else if  (strcmp(argv[i], "--deployer-address") == 0 || strcmp(argv[i], "-da") == 0) {
            input_deployer_address = argv[i + 1];
            i += 2;
        } else if  (strcmp(argv[i], "--work-scale") == 0 || strcmp(argv[i], "-w") == 0) {
            GRID_SIZE = 1U << atoi(argv[i + 1]);
            i += 2;
        } else if (strcmp(argv[i], "--prefix") == 0 || strcmp(argv[i], "-p") == 0) {
            input_prefix = argv[i + 1];
            // Convert input_prefix to lowercase
            for (char* p = input_prefix; *p; ++p) {
                *p = tolower(*p);
            }
            i += 2;
        } else if (strcmp(argv[i], "--suffix") == 0 || strcmp(argv[i], "-s") == 0) {
            input_suffix = argv[i + 1];
            // Convert input_suffix to lowercase
            for (char* p = input_suffix; *p; ++p) {
                *p = tolower(*p);
            }
            i += 2;
        } else {
            i++;
        }
    }

    // Debug print input_prefix and input_suffix after argument parsing, before validation/initialization
    if (input_prefix) {
        printf("[DEBUG] Input prefix: %s (length: %zu)\n", input_prefix, strlen(input_prefix));
    }
    if (input_suffix) {
        printf("[DEBUG] Input suffix: %s (length: %zu)\n", input_suffix, strlen(input_suffix));
    }

    if (num_devices == 0) {
        printf("[DEBUG] Exiting early due to no devices specified\n");
        printf("No devices were specified\n");
        return 1;
    }

    if (score_method == -1) {
        printf("Scoring method was not specified. Defaulting to 2. Mode: %d\n", mode);
    }

    if (mode == 2 && !input_file) {
        printf("[DEBUG] Exiting early due to missing contract bytecode for --contract2\n");
        printf("You must specify contract bytecode when using --contract2\n");
        return 1;
    }

    if ((mode == 2 || mode == 3) && !input_address) {
        printf("[DEBUG] Exiting early due to missing origin address for --contract2/3\n");
        printf("You must specify an origin address when using --contract2\n");
        return 1;
    } else if ((mode == 2 || mode == 3) && strlen(input_address) != 40 && strlen(input_address) != 42) {
        printf("[DEBUG] Exiting early due to origin address wrong length\n");
        printf("The origin address must be 40 characters long\n");
        return 1;
    }

    if ((mode == 2 || mode == 3) && !input_deployer_address) {
        printf("[DEBUG] Exiting early due to missing deployer address for --contract3\n");
        printf("You must specify a deployer address when using --contract3\n");
        return 1;
    }

    // If either prefix or suffix is set, switch to prefix/suffix scoring method
    if ((input_prefix && strlen(input_prefix) > 0) || (input_suffix && strlen(input_suffix) > 0)) {
        score_method = 2;
    }

    // Debug print the chosen score_method after all logic is finalized
    printf("[DEBUG] Score method selected: %d\n", score_method);



    for (int i = 0; i < num_devices; i++) {
        hipError_t e = hipSetDevice(device_ids[i]);
        if (e != hipSuccess) {
            printf("[DEBUG] Exiting early due to hipSetDevice failure for device %d\n", device_ids[i]);
            printf("Could not detect device %d\n", device_ids[i]);
            return 1;
        }
    }

    #define nothex(n) ((n < 48 || n > 57) && (n < 65 || n > 70) && (n < 97 || n > 102))
    _uint256 bytecode_hash;
    if (mode == 2 || mode == 3) {
        std::ifstream infile(input_file, std::ios::binary);
        if (!infile.is_open()) {
            printf("[DEBUG] Exiting early due to failed to open bytecode file\n");
            printf("Failed to open the bytecode file.\n");
            return 1;
        }
        
        int file_size = 0;
        {
            infile.seekg(0, std::ios::end);
            std::streampos file_size_ = infile.tellg();
            infile.seekg(0, std::ios::beg);
            file_size = file_size_ - infile.tellg();
        }

        if (file_size & 1) {
            printf("[DEBUG] Exiting early due to invalid bytecode file size\n");
            printf("Invalid bytecode in file.\n");
            return 1;
        }

        uint8_t* bytecode = new uint8_t[24576];
        if (bytecode == 0) {
            printf("[DEBUG] Exiting early due to error allocating memory for bytecode\n");
            printf("Error while allocating memory. Perhaps you are out of memory?");
            return 1;
        }

        char byte[3];
        byte[2] = '\0';
        bool prefix = false;
        for (int i = 0; i < (file_size >> 1); i++) {
            infile.read((char*)&byte, 2);
            if (i == 0) {
                prefix = byte[0] == '0' && byte[1] == 'x';
                if ((file_size >> 1) > (prefix ? 24577 : 24576)) {
                    printf("[DEBUG] Exiting early due to invalid bytecode in file (too long)\n");
                    printf("Invalid bytecode in file.\n");
                    delete[] bytecode;
                    return 1;
                }
                if (prefix) { continue; }
            }

            if (nothex(byte[0]) || nothex(byte[1])) {
                printf("[DEBUG] Exiting early due to invalid bytecode hex in file\n");
                printf("Invalid bytecode in file.\n");
                delete[] bytecode;
                return 1;
            }

            bytecode[i - prefix] = (uint8_t)strtol(byte, 0, 16);
        }    
        bytecode_hash = cpu_full_keccak(bytecode, (file_size >> 1) - prefix);
        delete[] bytecode;
    }

    Address origin_address;
    if (mode == 2 || mode == 3) {
        if (strlen(input_address) == 42) {
            input_address += 2;
        }
        char substr[9];
        substr[8] = '\0';

        #define round(i, offset) \
        strncpy(substr, input_address + offset * 8, 8); \
        if (nothex(substr[0]) || nothex(substr[1]) || nothex(substr[2]) || nothex(substr[3]) || nothex(substr[4]) || nothex(substr[5]) || nothex(substr[6]) || nothex(substr[7])) { \
            printf("[DEBUG] Exiting early due to invalid origin address\n"); \
            printf("Invalid origin address.\n"); \
            return 1; \
        } \
        origin_address.i = strtoull(substr, 0, 16);

        round(a, 0)
        round(b, 1)
        round(c, 2)
        round(d, 3)
        round(e, 4)

        #undef round
    }

    Address deployer_address;
    if (mode == 3) {
        if (strlen(input_deployer_address) == 42) {
            input_deployer_address += 2;
        }
        char substr[9];
        substr[8] = '\0';

        #define round(i, offset) \
        strncpy(substr, input_deployer_address + offset * 8, 8); \
        if (nothex(substr[0]) || nothex(substr[1]) || nothex(substr[2]) || nothex(substr[3]) || nothex(substr[4]) || nothex(substr[5]) || nothex(substr[6]) || nothex(substr[7])) { \
            printf("[DEBUG] Exiting early due to invalid deployer address\n"); \
            printf("Invalid deployer address.\n"); \
            return 1; \
        } \
        deployer_address.i = strtoull(substr, 0, 16);

        round(a, 0)
        round(b, 1)
        round(c, 2)
        round(d, 3)
        round(e, 4)

        #undef round
    }
    #undef nothex


    std::array<char, 64> prefix_copy{};
    std::array<char, 64> suffix_copy{};
    if (input_prefix && strlen(input_prefix) > 0) {
        std::strncpy(prefix_copy.data(), input_prefix, prefix_copy.size() - 1);
        prefix_copy.back() = '\0';
    }
    if (input_suffix && strlen(input_suffix) > 0) {
        std::strncpy(suffix_copy.data(), input_suffix, suffix_copy.size() - 1);
        suffix_copy.back() = '\0';
    }

    std::vector<std::thread> threads;
    uint64_t global_start_time = milliseconds();
    for (int i = 0; i < num_devices; i++) {
        std::thread th(host_thread, device_ids[i], i, score_method, mode, origin_address, deployer_address, bytecode_hash, prefix_copy.data(), suffix_copy.data());
        threads.push_back(move(th));
    }

    std::vector<double> speeds(num_devices, 0.0);
    while(true) {
        message_queue_mutex.lock();
        if (message_queue.empty()) {
            message_queue_mutex.unlock();
            std::this_thread::sleep_for(std::chrono::milliseconds(500));
            continue;
        }

        Message m = message_queue.front();
        message_queue.pop();
        message_queue_mutex.unlock();

        int device_index = m.device_index;

        if (m.status == 0) {
            if (device_index >= 0 && device_index < static_cast<int>(speeds.size())) {
                speeds[device_index] = m.speed;
            }
            printf("\r");
            if (m.results_count != 0) {
                Address* addresses = new Address[m.results_count];
                for (int i = 0; i < m.results_count; i++) {

                    if (mode == 0) {
                        CurvePoint p = cpu_point_multiply(G, m.results[i]);
                        addresses[i] = cpu_calculate_address(p.x, p.y);
                    } else if (mode == 1) {
                        CurvePoint p = cpu_point_multiply(G, m.results[i]);
                        addresses[i] = cpu_calculate_contract_address(cpu_calculate_address(p.x, p.y));
                    } else if (mode == 2) {
                        addresses[i] = cpu_calculate_contract_address2(origin_address, m.results[i], bytecode_hash);
                    } else if (mode == 3) {
                        _uint256 salt = cpu_calculate_create3_salt(origin_address, m.results[i]);
                        Address proxy = cpu_calculate_contract_address2(deployer_address, salt, bytecode_hash);
                        addresses[i] = cpu_calculate_contract_address(proxy, 1);
                    }
                }

                for (int i = 0; i < m.results_count; i++) {
                    _uint256 k = m.results[i];
                    int score = m.scores[i];
                    Address a = addresses[i];
                    uint64_t time = (m.time - global_start_time) / 1000;

                    if (mode == 0 || mode == 1) {
                        printf("Elapsed: %06u Score: %02u Private Key: 0x%08x%08x%08x%08x%08x%08x%08x%08x Address: 0x%08x%08x%08x%08x%08x\n", (uint32_t)time, score, k.a, k.b, k.c, k.d, k.e, k.f, k.g, k.h, a.a, a.b, a.c, a.d, a.e);
                    } else if (mode == 2 || mode == 3) {
                        printf("Elapsed: %06u Score: %02u Salt: 0x%08x%08x%08x%08x%08x%08x%08x%08x Address: 0x%08x%08x%08x%08x%08x\n", (uint32_t)time, score, k.a, k.b, k.c, k.d, k.e, k.f, k.g, k.h, a.a, a.b, a.c, a.d, a.e);
                    } else {
                        printf("Final mode%d", mode);
                    }
                }

                delete[] addresses;
                delete[] m.results;
                delete[] m.scores;
            }
            print_speeds(num_devices, device_ids, speeds);
            fflush(stdout);
        } else if (m.status == 1) {
            const int dev_id = (device_index >= 0 && device_index < num_devices) ? device_ids[device_index] : -1;
            printf("\rCuda error %d on device %d. Device will halt work.\n", m.error, dev_id);
            print_speeds(num_devices, device_ids, speeds);
            fflush(stdout);
        } else if (m.status == 11) {
            const int dev_id = (device_index >= 0 && device_index < num_devices) ? device_ids[device_index] : -1;
            printf("\rError from BCryptGenRandom. Device %d will halt work.", dev_id);
            print_speeds(num_devices, device_ids, speeds);
            fflush(stdout);
        } else if (m.status == 12) {
            const int dev_id = (device_index >= 0 && device_index < num_devices) ? device_ids[device_index] : -1;
            printf("\rError while reading from /dev/urandom. Device %d will halt work.", dev_id);
            print_speeds(num_devices, device_ids, speeds);
            fflush(stdout);
        } else if (m.status == 13) {
            const int dev_id = (device_index >= 0 && device_index < num_devices) ? device_ids[device_index] : -1;
            printf("\rError while opening /dev/urandom. Device %d will halt work.", dev_id);
            print_speeds(num_devices, device_ids, speeds);
            fflush(stdout);
        } else if (m.status == 100) {
            const int dev_id = (device_index >= 0 && device_index < num_devices) ? device_ids[device_index] : -1;
            printf("\rError while allocating memory. Perhaps you are out of memory? Device %d will halt work.", dev_id);
        }
    }
}